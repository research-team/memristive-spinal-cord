#include "hip/hip_runtime.h"
#include "core.cu"
#include "structs.h"

int TEST;
double E2F_coef;
double V0v2F_coef;
double EES_test_stregth;
double QUADRU_Ia;
const char layers = 5;      // number of OM layers (5 is default)
const char CV_number = 6;
const char extra_layers = 0 + layers;
int ees_fr = 40;      // frequency of EES

void init_network() {
	/**
	 * todo
	 */
	 /* test
	auto ees = form_group("EES", 1, GENERATOR);
	auto inter = form_group("inter", 1, INTER);
	auto afferent = form_group("afferent", 1, AFFERENTS);
	auto moto = form_group("moto", 1, MOTO);

	add_generator(ees, 0, sim_time, 40);

	conn_generator(ees, inter, 10, 0.05, 1, -1);
	conn_generator(ees, afferent, 1, 0.5, 1, -1);

	connect_fixed_indegree(inter, moto, 2, 7, 1, -1);
	connect_fixed_indegree(afferent, moto, 2, 7, 1, -1);

	save({inter, afferent, moto});

	return;
	  */
	string name;
	vector<Group> E, CV, L0, L1, L2E, L2F, L3, IP_E, IP_F, gen_C, C_0, V0v;
	// generators
	auto ees = form_group("EES", 1, GENERATOR);
	auto MOTO_NOISE = form_group("MOTO_NOISE", 1, GENERATOR);
	// 
	for(int layer = 0; layer < CV_number; ++layer) {
		name = to_string(layer + 1);
		gen_C.push_back(form_group("C" + name, 1, GENERATOR));
	}
	// EXTRA layers
	for(int layer = layers; layer < extra_layers; ++layer) {
		name = to_string(layer + 1);
		gen_C.push_back(form_group("C" + name, 1, GENERATOR));
	}

	for(int step = 0; step < step_number; ++step) {
		name = to_string(step);
		C_0.push_back(form_group("C_0_step_" + name, 1, GENERATOR));
		V0v.push_back(form_group("V0v_step_" + name, 1, GENERATOR));
	}
	//
	auto OM1_0E = form_group("OM1_0E");
	auto OM1_0F = form_group("OM1_0F");
	// OM groups by layer
	for(int layer = 0; layer < layers; ++layer) {
		name = to_string(layer + 1);
		L0.push_back(form_group("OM" + name + "_0"));
		L1.push_back(form_group("OM" + name + "_1"));
		L2E.push_back(form_group("OM" + name + "_2E"));
		L2F.push_back(form_group("OM" + name + "_2F"));
		L3.push_back(form_group("OM" + name + "_3"));
	}
	// EXTRA OM groups by layer
	for(int layer = layers; layer < extra_layers; ++layer) {
		name = to_string(layer + 1);
		L0.push_back(form_group("OM" + name + "_0"));
		L1.push_back(form_group("OM" + name + "_1"));
		L2E.push_back(form_group("OM" + name + "_2E"));
		L2F.push_back(form_group("OM" + name + "_2F"));
		L3.push_back(form_group("OM" + name + "_3"));
	}

	//
	for(int layer = 0; layer < CV_number; ++layer) {
		name = to_string(layer + 1);
		E.push_back(form_group("E" + name, 50, AFFERENTS));
		CV.push_back(form_group("CV_" + name, 50, AFFERENTS));
	}

	for(int layer = 0; layer < extra_layers; ++layer) {
		name = to_string(layer + 1);
		IP_E.push_back(form_group("IP_E_" + name));
		IP_F.push_back(form_group("IP_F_" + name));
	}
	// afferents
	auto Ia_aff_E = form_group("Ia_aff_E", 120, AFFERENTS);
	auto Ia_aff_F = form_group("Ia_aff_F", 120, AFFERENTS);
	// motoneurons
	auto mns_E = form_group("mns_E", 210, MOTO);
	auto mns_F = form_group("mns_F", 180, MOTO);
	// muscle fibers
	auto muscle_E = form_group("muscle_E", 210 * 50, MUSCLE, 3); // 150 * 210
	auto muscle_F = form_group("muscle_F", 180 * 50, MUSCLE, 3); // 100 * 180
	// reflex arc E
	auto Ia_E = form_group("Ia_E", neurons_in_ip);
	auto iIP_E = form_group("iIP_E", neurons_in_ip);
	auto R_E = form_group("R_E");
	// reflex arc F
	auto Ia_F = form_group("Ia_F", neurons_in_ip);
	auto iIP_F = form_group("iIP_F", neurons_in_ip);
	auto R_F = form_group("R_F");

	// create EES generator
	add_generator(ees, 25, sim_time, ees_fr);

	add_generator(MOTO_NOISE, 0, sim_time, 200);

	// create CV generators (per step)
	for (int layer = 0; layer < CV_number + TEST; ++layer) {
		for (int step_index = 0; step_index < step_number; ++step_index) {
			normal_distribution<double> freq_distr(cv_fr, cv_fr / 10);
			double start = 25 + skin_time * layer + step_index * (skin_time * CV_number + 25 * slices_flexor);

			double end = start + skin_time - 3; // remove merging CV
			add_generator(gen_C[layer], start, end, freq_distr(rand_gen));
		}
		printf("step\n");
	}

	// EXTRA create CV generators (per step)
	for (int layer = layers; layer < extra_layers + TEST; ++layer) {
		for (int step_index = 0; step_index < step_number; ++step_index) {
			normal_distribution<double> freq_distr(cv_fr, cv_fr / 10);
			double start = 25 + skin_time * (layer - 4) + step_index * (skin_time * CV_number + 25 * slices_flexor);
//			double start = 25 + skin_time * (layer) + step_index * (skin_time * CV_number + 25 * slices_flexor);
			double end = start + skin_time - 3; // remove merging CV
			add_generator(gen_C[layer], start, end, freq_distr(rand_gen));
		}
		printf("step\n");
	}

	// create C_0 and V0v generators (per step)
	for (int step_index = 0; step_index < step_number; ++step_index) {
		// freq = 200 (interval = 5ms), count = 125 / interval. Duration = count * interval = 125
		double start = 25 + skin_time * slices_extensor + step_index * (skin_time * slices_extensor + 25 * slices_flexor);
		double end = start + 25 * slices_flexor - 7;
		add_generator(C_0[step_index], start, end, cv_fr);
		// V0v
		start = 20 + skin_time * slices_extensor + step_index * (skin_time * slices_extensor + 25 * slices_flexor);
		end = start + 75; // 75
		add_generator(V0v[step_index], start, end, cv_fr);
	}

	// extensor
	createmotif(OM1_0E, L1[0], L2E[0], L3[0]);
	for(int layer = 1; layer < layers; ++layer)
		createmotif(L0[layer], L1[layer], L2E[layer], L3[layer]);
	// EXTRA layers extensor
	for(int layer = layers; layer < extra_layers; ++layer)
		createmotif(L0[layer], L1[layer], L2E[layer], L3[layer]);
	
	// FLEXOR
	createmotif_flexor(OM1_0F, L1[0], L2F[0], L3[0]);
	for(int layer = 1; layer < layers; ++layer)
		createmotif_flexor(L0[layer], L1[layer], L2F[layer], L3[layer]);
	for(int layer = 1; layer < layers; ++layer)
		connect_fixed_indegree(L2F[layer - 1], L2F[layer], 3 + 1, 0.2, 50, 2);
	// EXTRA layers FLEXOR
	for(int layer = layers; layer < extra_layers; ++layer)
		connect_fixed_indegree(L2F[layer - 1], L2F[layer], 3 + 1, 0.2, 50, 2);
	
	//
	connect_fixed_indegree(E[0], OM1_0F, 3, 0.00025 * E2F_coef, 50, 3);
	for(int step = 0; step < step_number; ++step) {
		connect_fixed_indegree(V0v[step], OM1_0F, 3, 0.75 * V0v2F_coef, 50, 5);
	}
	// between delays via excitatory pools
	// extensor

	/// !!!!!
	for(int layer = 1; layer < extra_layers; ++layer) {
		connect_fixed_indegree(E[layer - 1], E[layer], 2 + 1, 0.75); // 4.75
	}
	// connect E (from EES)
	connect_fixed_indegree(E[0], OM1_0E, 2 + 0.5, 0.005 * 0.8 * E_coef, 50, 4); // 0.00040 - 0.00047
	for(int layer = 1; layer < extra_layers; ++layer) {
		connect_fixed_indegree(E[layer], L0[layer], 2 + 0.5, 0.005 * 0.8 * E_coef, 50, 4); // 0.00048 * 0.4, 1.115
	}

	// E inhibitory projections (via 3rd core)
	/*
	for (int layer = 0; layer < layers - 1; ++layer) {
		if (layer >= 3) {
			for (int i = layer + 3; i < layers + 1 + TEST; ++i) {
				printf("C index %d, OM%d_3 (layer > 3)\n", i, layer);
				connect_fixed_indegree(gen_C[i], L3[layer], 1, 1.95);
			}
		} else {
			for (int i = layer + 2; i < layers + 1 + TEST; ++i) {
				printf("C index %d, OM%d_3 (else)\n", i, layer);
				connect_fixed_indegree(gen_C[i], L3[layer], 1, 1.95);
			}
		}
	}*/
	for (int layer = 2; layer < layers + 1; ++layer) {
		if (layer > 3) {
			for (int i = 0; i < layer - 2 + TEST; ++i) {
				printf("C index %d, OM%d_3 (layer > 3)\n", i, layer);
				connect_fixed_indegree(gen_C[layer], L3[i], 1, 1.95);
			}
		} else {
			for (int i = 0; i < layer - 1 + TEST; ++i) {
				printf("C index %d, OM%d_3 (else)\n", i, layer);
				connect_fixed_indegree(gen_C[layer], L3[i], 1, 1.95);
			}
		}
	}
	// EXTRA layers
	for (int layer = layers; layer < extra_layers; ++layer) {
		printf("C index %d, OM%d_3 (else)\n", layer, layer);
		connect_fixed_indegree(gen_C[layer - 3], L3[layer], 0.1, 0.1);
	}

	conn_generator(ees, Ia_aff_E, 1, 2.5 * EES_test_stregth);
	conn_generator(ees, Ia_aff_F, 1, 2.5 * EES_test_stregth);
	conn_generator(ees, E[0], 3, 1.0 * EES_test_stregth, 50, -1); // NORMAL
	///conn_generator(Iagener_E, Ia_aff_E, 1, 0.0001, 5);
	///conn_generator(Iagener_F, Ia_aff_F, 1, 0.0001, 5);
	// TODO motifs disable weight inh
	connect_fixed_indegree(Ia_aff_E, mns_E, 1.0, 0.045 * QUADRU_Ia); // was 1.5ms 0.045
	connect_fixed_indegree(Ia_aff_F, mns_F, 2.0, 0.006);

	connect_fixed_outdegree_MUSCLE(mns_E, muscle_E, 1.2, 0.11, 45); // 2.0
	connect_fixed_outdegree_MUSCLE(mns_F, muscle_F, 1.2, 0.38, 45); // 2.0

	connect_fixed_outdegree_MUSCLE(MOTO_NOISE, mns_E, 5, 0.05, 50, 5);
	connect_fixed_outdegree_MUSCLE(MOTO_NOISE, mns_F, 5, 0.5, 50, 5);

	if (layers >= 1)
		connect_fixed_indegree(gen_C[0], Ia_aff_E, 2, -0.4, 50, 3);
	if (layers >= 2)
		connect_fixed_indegree(gen_C[1], Ia_aff_E, 2, -0.2, 50, 3);
	// if (layers >= 3)
		// connect_fixed_indegree(CV[3], Ia_aff_E, 1, -0.1, 50, 3);
	if (layers >= 4)
		connect_fixed_indegree(gen_C[4], Ia_aff_E, 2, -0.2, 50, 3);
	if (layers >= 5)
		connect_fixed_indegree(gen_C[5], Ia_aff_E, 2, -0.4, 50, 3);

	// connect_fixed_outdegree_MUSCLE(gen_C[0], mns_E, 4, -2.5, 100, 5); // 2.0
	// connect_fixed_outdegree_MUSCLE(gen_C[1], mns_E, 4, -1, 100, 5); // 2.0
	// connect_fixed_outdegree_MUSCLE(gen_C[4], mns_E, 4, -1, 100, 5); // 2.0
	// connect_fixed_outdegree_MUSCLE(gen_C[5], mns_E, 4, -2.5, 100, 5); // 2.0

	// connect_fixed_outdegree_MUSCLE(MOTO_NOISE, muscle_E, 5, 0.1, 50, 5);
	// connect_fixed_outdegree_MUSCLE(MOTO_NOISE, muscle_F, 5, 0.3, 50, 5);

	// IP
	for (int layer = 0; layer < layers; ++layer) {
		// Extensor
		connect_fixed_indegree(L2E[layer], IP_E[layer], 2, 0.005, 500, 5); // 2.5
		connect_fixed_indegree(IP_E[layer], mns_E, 2, 0.0045, 500, 5); // 0.005
		if (layer > 3)
			connect_fixed_indegree(IP_E[layer], Ia_aff_E, 1, -0.0002 *layer);
		else
			connect_fixed_indegree(IP_E[layer], Ia_aff_E, 1, -0.0001);
		// Flexor
		connect_fixed_indegree(L2F[layer], IP_F[layer], 2, 0.001, 500, 5); // 2.5
		connect_fixed_indegree(IP_F[layer], mns_F, 3, 0.004, 100, 5); // 2.75 0.125 0.2
	}

	for (int layer = layers; layer < extra_layers; ++layer) {
		// Extensor
		connect_fixed_indegree(L2E[layer], IP_E[layer], 2, 0.005, 500, 5); // 2.5
		connect_fixed_indegree(IP_E[layer], mns_E, 2, 0.0045, 500, 5); // 0.005
		// Flexor
		connect_fixed_indegree(L2F[layer], IP_F[layer], 2, 0.001, 500, 5); // 2.5
		connect_fixed_indegree(IP_F[layer], mns_F, 3, 0.004, 100, 5); // 2.75 0.125 0.2
	}

	// skin inputs
	for (int layer = 0; layer < CV_number + TEST; ++layer)
		connect_fixed_indegree(gen_C[layer], CV[layer], 2, 0.15 * cv_coef);
	// CV
	double TESTCOEF = 30.0;
	double T_coef = 1.0;
	// OM1
	connect_fixed_indegree(CV[0], OM1_0E, 2 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	connect_fixed_indegree(CV[1], OM1_0E, 2 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	// OM2
	if (layers >= 1) {
		connect_fixed_indegree(CV[0], L0[1], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[1], L0[1], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		if (layers >= 2) connect_fixed_indegree(CV[2], L0[1], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	}
	// OM3
	if (layers >= 2) {
		connect_fixed_indegree(CV[0], L0[2], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[1], L0[2], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[2], L0[2], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		if (layers >= 3) connect_fixed_indegree(CV[3], L0[2], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		if (layers >= 4) connect_fixed_indegree(CV[4], L0[2], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	}
	// OM4
	if (layers >= 3) {
		connect_fixed_indegree(CV[1], L0[3], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[2], L0[3], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[3], L0[3], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		if (layers >= 4) connect_fixed_indegree(CV[4], L0[3], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		if (layers >= 5) connect_fixed_indegree(CV[5], L0[3], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	}
	// OM5
	if (layers >= 4) {
		connect_fixed_indegree(CV[1], L0[4], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[2], L0[4], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[3], L0[4], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[4], L0[4], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		if (layers >= 5) connect_fixed_indegree(CV[5], L0[4], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	}

	// OM6
	if (extra_layers > 5) {
		connect_fixed_indegree(CV[1], L0[5], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[2], L0[5], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[3], L0[5], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[4], L0[5], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[5], L0[5], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
//		if (extra_layers >= 6) connect_fixed_indegree(CV[6], L0[5], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	}

	// OM7
	if (extra_layers > 6) {
		connect_fixed_indegree(CV[1], L0[6], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[2], L0[6], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[3], L0[6], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[4], L0[6], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[5], L0[6], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
		connect_fixed_indegree(CV[6], L0[6], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
//		if (extra_layers >= 7) connect_fixed_indegree(CV[7], L0[6], 3 + T_coef, 0.00045 * cv_coef * TESTCOEF * 0.3, 50, 3);
	}

	// C=1 Extensor
	for (int layer = 0; layer < layers; ++layer)
		connect_fixed_indegree(IP_E[layer], iIP_E, 1, 0.001);

	for (int layer = 0; layer < layers + TEST; ++layer) {
		connect_fixed_indegree(CV[layer], iIP_E, 4, 1); // for what CV and Gen_C
		connect_fixed_indegree(gen_C[layer], iIP_E, 1, 1);
	}
	connect_fixed_indegree(iIP_E, OM1_0F, 0.1, -0.001);

	for (int layer = 0; layer < layers; ++layer) {
		connect_fixed_indegree(iIP_E, L2F[layer], 2, -0.4); //0.8 0.6
		connect_fixed_indegree(iIP_F, L2E[layer], 2, -0.5);
	}
	//
	connect_fixed_indegree(iIP_E, Ia_aff_F, 1, -1.2);
	connect_fixed_indegree(iIP_E, mns_F, 4, -0.0315); // 0.08 delay 0.1

	for (int layer = 0; layer < layers; ++layer) {
		connect_fixed_indegree(iIP_E, IP_F[layer], 4.5, -0.005); // 0.1
		connect_fixed_indegree(IP_F[layer], iIP_F, 1, 0.0001);
		connect_fixed_indegree(iIP_F, IP_E[layer], 1, -0.08); // -0.1
	}
	// C=0 Flexor
	connect_fixed_indegree(iIP_F, iIP_E, 1, -0.5);
	connect_fixed_indegree(iIP_F, Ia_aff_E, 1, -3.5);
	connect_fixed_indegree(iIP_F, mns_E, 1, -0.35); //0.5
	for(int step = 0; step < step_number; ++step) {
		connect_fixed_indegree(C_0[step], iIP_F, 1, 0.8);
	}
	// reflex arc
	connect_fixed_indegree(iIP_E, Ia_E, 1, 0.001);
	connect_fixed_indegree(Ia_aff_E, Ia_E, 1, 0.008);
	connect_fixed_indegree(mns_E, R_E, 1, 0.00015);
	connect_fixed_indegree(Ia_E, mns_F, 0.1, -0.002);
//	connect_fixed_indegree(R_E, mns_E, 1, -0.00015);
	connect_fixed_indegree(R_E, Ia_E, 1, -0.001);
	//
	connect_fixed_indegree(iIP_F, Ia_F, 1, 0.001);
	connect_fixed_indegree(Ia_aff_F, Ia_F, 1, 0.008);
	connect_fixed_indegree(mns_F, R_F, 1, 0.00015);
//	connect_fixed_indegree(Ia_F, mns_E, 1, -0.08);

	connect_fixed_indegree(R_F, mns_F, 0.1, -0.00015);
	connect_fixed_indegree(R_F, Ia_F, 1, -0.001);
	// todo C_0

	//
	connect_fixed_indegree(R_E, R_F, 1, -0.04);
	connect_fixed_indegree(R_F, R_E, 1, -0.04);
	connect_fixed_indegree(Ia_E, Ia_F, 1, -0.08);
	connect_fixed_indegree(Ia_F, Ia_E, 1, -0.08);
	connect_fixed_indegree(iIP_E, iIP_F, 3, -0.04); // delay 1
	connect_fixed_indegree(iIP_F, iIP_E, 1, -0.04);

	save({muscle_E, muscle_F});
//	 save(all_groups);
}


void simulate(int test_index) {
	/**
	 *
	 */
	random_device r;
	default_random_engine rand_gen(r());
	uniform_real_distribution<double> delay_distr(2, 4);

	// init structs (CPU)
	States *S = (States *)malloc(sizeof(States));
	Parameters *P = (Parameters *)malloc(sizeof(Parameters));
	Neurons *N = (Neurons *)malloc(sizeof(Neurons));
	Synapses *synapses = (Synapses *)malloc(sizeof(Synapses));
	Generators *G = (Generators *)malloc(sizeof(Generators));

	// create neurons and their connectomes
	init_network();
	// note: important
	vector_nrn_start_seg.push_back(NRNS_AND_SEGS);

	// allocate generators into the GPU
	unsigned int gens_number = vec_spike_each_step.size();
	G->nrn_id = init_gpu_arr(vec_nrn_id);
	G->time_end = init_gpu_arr(vec_time_end);
	G->freq_in_steps = init_gpu_arr(vec_freq_in_steps);
	G->spike_each_step = init_gpu_arr(vec_spike_each_step);
	G->size = gens_number;

	// allocate static parameters into the GPU
	P->nrn_start_seg = init_gpu_arr(vector_nrn_start_seg);
	P->models = init_gpu_arr(vector_models);
	P->Cm = init_gpu_arr(vector_Cm);
	P->gnabar = init_gpu_arr(vector_gnabar);
	P->gkbar = init_gpu_arr(vector_gkbar);
	P->gl = init_gpu_arr(vector_gl);
	P->Ra = init_gpu_arr(vector_Ra);
	P->diam = init_gpu_arr(vector_diam);
	P->length = init_gpu_arr(vector_length);
	P->ena = init_gpu_arr(vector_ena);
	P->ek = init_gpu_arr(vector_ek);
	P->el = init_gpu_arr(vector_el);
	P->gkrect = init_gpu_arr(vector_gkrect);
	P->gcaN = init_gpu_arr(vector_gcaN);
	P->gcaL = init_gpu_arr(vector_gcaL);
	P->gcak = init_gpu_arr(vector_gcak);
	P->E_ex = init_gpu_arr(vector_E_ex);
	P->E_inh = init_gpu_arr(vector_E_inh);
	P->tau_exc = init_gpu_arr(vector_tau_exc);
	P->tau_inh1 = init_gpu_arr(vector_tau_inh1);
	P->tau_inh2 = init_gpu_arr(vector_tau_inh2);
	P->size = NRNS_NUMBER;

	// dynamic states of neuron (CPU arrays) and allocate them into the GPU
//	double *Vm; HANDLE_ERROR(hipHostMalloc((void**)&Vm, NRNS_AND_SEGS));
	auto *Vm = arr_init<double>(); S->Vm = init_gpu_arr(Vm);
	auto *n = arr_init<double>(); S->n = init_gpu_arr(n);
	auto *m = arr_init<double>(); S->m = init_gpu_arr(m);
	auto *h = arr_init<double>(); S->h = init_gpu_arr(h);
	auto *l = arr_init<double>(); S->l = init_gpu_arr(l);
	auto *s = arr_init<double>(); S->s = init_gpu_arr(s);
	auto *p = arr_init<double>(); S->p = init_gpu_arr(p);
	auto *hc = arr_init<double>(); S->hc = init_gpu_arr(hc);
	auto *mc = arr_init<double>(); S->mc = init_gpu_arr(mc);
	auto *cai = arr_init<double>(); S->cai = init_gpu_arr(cai);
	auto *I_Ca = arr_init<double>(); S->I_Ca = init_gpu_arr(I_Ca);
	auto *NODE_A = arr_init<double>(); S->NODE_A = init_gpu_arr(NODE_A);
	auto *NODE_B = arr_init<double>(); S->NODE_B = init_gpu_arr(NODE_B);
	auto *NODE_D = arr_init<double>(); S->NODE_D = init_gpu_arr(NODE_D);
	auto *const_NODE_D = arr_init<double>(); S->const_NODE_D = init_gpu_arr(const_NODE_D);
	auto *NODE_RHS = arr_init<double>(); S->NODE_RHS = init_gpu_arr(NODE_RHS);
	auto *NODE_RINV = arr_init<double>(); S->NODE_RINV = init_gpu_arr(NODE_RINV);
	auto *NODE_AREA = arr_init<double>(); S->NODE_AREA = init_gpu_arr(NODE_AREA);

//	int ext_size = NRNS_AND_SEGS * 2;
//	auto *EXT_A = arr_init<double>(ext_size); S->EXT_A = init_gpu_arr(EXT_A, ext_size);
//	auto *EXT_B = arr_init<double>(ext_size); S->EXT_B = init_gpu_arr(EXT_B, ext_size);
//	auto *EXT_D = arr_init<double>(ext_size); S->EXT_D = init_gpu_arr(EXT_D, ext_size);
//	auto *EXT_V = arr_init<double>(ext_size); S->EXT_V = init_gpu_arr(EXT_V, ext_size);
//	auto *EXT_RHS = arr_init<double>(ext_size); S->EXT_RHS = init_gpu_arr(EXT_RHS, ext_size);
	S->size = NRNS_AND_SEGS;
//	S->ext_size = ext_size;

	// special neuron's state (CPU) and allocate them into the GPU
	auto *tmp = arr_init<double>(NRNS_NUMBER);
	for (int i = 0; i < NRNS_NUMBER; ++i)
		tmp[i] = 0.0;

	auto *has_spike = arr_init<bool>(NRNS_NUMBER); N->has_spike = init_gpu_arr(has_spike, NRNS_NUMBER);
	auto *g_exc = arr_init<double>(NRNS_NUMBER); N->g_exc = init_gpu_arr(g_exc, NRNS_NUMBER);
	auto *g_inh_A = arr_init<double>(NRNS_NUMBER); N->g_inh_A = init_gpu_arr(g_inh_A, NRNS_NUMBER);
	auto *g_inh_B = arr_init<double>(NRNS_NUMBER); N->g_inh_B = init_gpu_arr(g_inh_B, NRNS_NUMBER);
	auto *spike_on = arr_init<bool>(NRNS_NUMBER); N->spike_on = init_gpu_arr(spike_on, NRNS_NUMBER);
	auto *factor = arr_init<double>(NRNS_NUMBER); N->factor = init_gpu_arr(factor, NRNS_NUMBER);
	auto *ref_time_timer = arr_init<unsigned int>(NRNS_NUMBER); N->ref_time_timer = init_gpu_arr(ref_time_timer, NRNS_NUMBER);
	auto *ref_time = arr_init<unsigned int>(NRNS_NUMBER);
	for (int i = 0; i < NRNS_NUMBER; ++i)
		ref_time[i] = ms_to_step(delay_distr(rand_gen));
	N->ref_time = init_gpu_arr(ref_time, NRNS_NUMBER);
	N->size = NRNS_NUMBER;

	// synaptic parameters
	unsigned int synapses_number = vector_syn_delay.size();
	synapses->syn_pre_nrn = init_gpu_arr(vector_syn_pre_nrn);
	synapses->syn_post_nrn = init_gpu_arr(vector_syn_post_nrn);
	synapses->syn_weight = init_gpu_arr(vector_syn_weight);
	synapses->syn_delay = init_gpu_arr(vector_syn_delay);
	synapses->syn_delay_timer = init_gpu_arr(vector_syn_delay_timer);
	synapses->size = synapses_number;

	// allocate structs to the device
	auto *dev_S = init_gpu_arr(S, 1);
	auto *dev_P = init_gpu_arr(P, 1);
	auto *dev_N = init_gpu_arr(N, 1);
	auto *dev_G = init_gpu_arr(G, 1);
	auto *dev_synapses = init_gpu_arr(synapses, 1);

	printf("Network: %d neurons (with segs: %d), %d synapses, %d generators\n",
	       NRNS_NUMBER, NRNS_AND_SEGS, synapses_number, gens_number);

	int THREADS = 32, BLOCKS = 10;

	hiprandState *devStates;
	HANDLE_ERROR(hipMalloc((void **)&devStates, NRNS_NUMBER * sizeof(hiprandState)));

	float time;
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// call initialisation kernel
	initialization_kernel<<<1, 1>>>(devStates, dev_S, dev_P, dev_N, -70.0);

	// the main simulation loop
	for (unsigned int sim_iter = 0; sim_iter < SIM_TIME_IN_STEPS; ++sim_iter) {
		if (sim_iter % 1000 == 0) {
			printf("%.2f%% is done\n", 100.0 * sim_iter / SIM_TIME_IN_STEPS);
		}
		/// KERNEL ZONE
		// deliver_net_events, synapse updating and neuron conductance changing kernel
		synapse_kernel<<<5, 256>>>(dev_N, dev_synapses);
		// updating neurons kernel
		neuron_kernel<<<BLOCKS, THREADS>>>(devStates, dev_S, dev_P, dev_N, dev_G, sim_iter);
		/// SAVE DATA ZONE
		/*
		// int streamSize = NRNS_AND_SEGS / 4;
		// int streamSizeTail = NRNS_AND_SEGS % 4;
		// for (int streamIndex = 0; streamIndex < nStreams; ++streamIndex) {
		// 	int offset = streamIndex * streamSize;
		// 	if (streamIndex == nStreams - 1)
		// 		offset += streamSizeTail;
		// 	hipMemcpyAsync(&Vm[offset], &S->Vm[offset], streamBytes, hipMemcpyDeviceToHost, hipMemcpyDeviceToHost, stream[streamIndex]);
		// 	hipMemcpyAsync(&g_exc[offset], &N->g_exc[offset], streamBytes, hipMemcpyDeviceToHost, hipMemcpyDeviceToHost, stream[streamIndex]);
		// 	hipMemcpyAsync(&g_inh_A[offset], &N->g_inh_A[offset], streamBytes, hipMemcpyDeviceToHost, hipMemcpyDeviceToHost, stream[streamIndex]);
		// 	hipMemcpyAsync(&g_inh_B[offset], &N->g_inh_B[offset], streamBytes, hipMemcpyDeviceToHost, hipMemcpyDeviceToHost, stream[streamIndex]);
		// 	hipMemcpyAsync(&has_spike[offset], &N->has_spike[offset], streamBytes, hipMemcpyDeviceToHost, hipMemcpyDeviceToHost, stream[streamIndex]);
		// }*/
		memcpyDtH(S->Vm, Vm, NRNS_AND_SEGS);
		memcpyDtH(N->g_exc, g_exc, NRNS_NUMBER);
		memcpyDtH(N->g_inh_A, g_inh_A, NRNS_NUMBER);
		memcpyDtH(N->g_inh_B, g_inh_B, NRNS_NUMBER);
		memcpyDtH(N->has_spike, has_spike, NRNS_NUMBER);
		// fill records arrays
		for (GroupMetadata& metadata : saving_groups) {
			copy_data_to(metadata, Vm, tmp, g_exc, g_inh_A, g_inh_B, has_spike, sim_iter);
		}
	}
	// properly ending work with GPU
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
	// todo optimize the code to free all GPU variables
	// HANDLE_ERROR(hipFree(S->Vm));
	// HANDLE_ERROR( hipHostFree( S->Vm ) );

	// stuff info
	printf("Elapsed GPU time: %d ms\n", (int) time);
	double Tbw = 12000 * pow(10, 6) * (128 / 8) * 2 / pow(10, 9);
	printf("Theoretical Bandwidth GPU (2 Ghz, 128 bit): %.2f GB/s\n", Tbw);

	// save the data into the current folder
	save_result(test_index);
}

int main(int argc, char **argv) {
	enum modes {air, toe, plt, quadru, normal, qpz, str, s6, s13, s21};
	//
	int iter = atoi(argv[1]);
	int arg_exp = atoi(argv[2]);
	int arg_hz = atoi(argv[3]);

	modes bws, pharma, speed;

	if (arg_exp == 0) {
		printf("PLT 13.5\n");
		bws = plt;     
		pharma = normal;  
		speed = s13;
	} else if (arg_exp == 1) {
		printf("TOE 13.5\n");
		bws = toe;
		pharma = normal;
		speed = s13;
	} else if (arg_exp == 2) {
		printf("AIR 13.5\n");
		bws = air;
		pharma = normal;
		speed = s13;
	} else if (arg_exp == 3) {
		printf("QUADRU 13.5\n");
		bws = quadru;     
		pharma = normal;  
		speed = s13;    
	} else if (arg_exp == 4) {
		printf("QPZ 13.5\n");
		bws = plt; 
		pharma = qpz;
		speed = s13; 
	} else if (arg_exp == 5) {
		printf("STR 13.5\n");
		bws = plt; 
		pharma = str;
		speed = s13;  
	} else if (arg_exp == 6) {
		printf("PLT 21\n");
		bws = plt;     
		pharma = normal;
		speed = s21;
	} else if (arg_exp == 7) {
		printf("PLT 6\n");
		bws = plt;     
		pharma = normal;
		speed = s6;    
	}

	step_number = 11;
	
	TEST = 0;
	E2F_coef = 1;
	V0v2F_coef = 1;
	QUADRU_Ia = 1;
	EES_test_stregth = 1.0;
	ees_fr = arg_hz;

	// speed modes
	switch(speed) {
		case s6:
			skin_time = 125;
			break;
		case s13:
			skin_time = 50;
			break;
		case s21:
			skin_time = 25;
			break;
		default:
			exit(-1);
	}
	// BWS modes
	switch(bws) {
		case air:
			TEST = -1;
			skin_time = 25;
			cv_coef = 0.043; // 037
			E_coef = 0.05;
			slices_extensor = 5;
			slices_flexor = 4;
			E2F_coef = 0;
			V0v2F_coef = 0;
			break;
		case toe:
			TEST = -2;
			cv_coef = 0.05;
			E_coef = 0.05;
			slices_extensor = 4;
			slices_flexor = 4;
			E2F_coef = 8;
			V0v2F_coef = 0;
			break;
		case plt: //!
			QUADRU_Ia = 1.0;
			cv_coef = 0.07;			// cv_coef = 0.0615;	gut 0.08
			E_coef = 0.05;			// 	E_coef = 0.052;
			slices_extensor = 6;	// 	slices_extensor = 6;		
			slices_flexor = 5;		// 	slices_flexor = 5;	
			E2F_coef = 8;			// 	E2F_coef = 8;
			V0v2F_coef = 0.001;		// 	V0v2F_coef = 0.001;	
			break;
		case quadru:
			QUADRU_Ia = 1.0;
			cv_coef = 0.036; // 0.042
			E_coef = 0.039; // 0.045
			slices_extensor = 6;
			slices_flexor = 7;
			E2F_coef = 8;
			V0v2F_coef = 0.001;
			break;
		default:
			exit(-1);
	}
	// pharma modes
	switch(pharma) {
		case normal:
			break;
		case qpz:
			QUADRU_Ia = 1.5;
			cv_coef = 0.15; // 0.05 1ю15
			E_coef = 0.17; // 0.07  3ю0
			V0v2F_coef = 0.001;
			break;
		case str:
			QUADRU_Ia = 0.8;
			str_flag = true;
			V0v2F_coef = 0.001;
			break;
		default:
			exit(-1);
	}

	// remove
	// E_coef = 0.0025;
	// cv_coef = 0.069;
	// testing a weak EES and increased a CV strength
	// cv_coef = 0.09;
	// EES_test_stregth = 0.001;

	// one_step_time = slices_extensor * skin_time + 25 * slices_flexor;
	int fr = 1000 / ees_fr;
	one_step_time = (slices_extensor * skin_time + 25 * slices_flexor) / fr * fr;
	printf("STEP LENGTH %d\n", one_step_time);
	sim_time = 25 + one_step_time * step_number;
	SIM_TIME_IN_STEPS = (unsigned int)(sim_time / dt);  // [steps] converted time into steps

	// init the device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, dev));
	printf("%s struct of array at device %d: %s \n", argv[0], dev, deviceProp.name);
	HANDLE_ERROR(hipSetDevice(dev));
	
	printf("%d\n", arg_exp * 10 + iter);
	// the main body of simulation
	//simulate(arg_exp * 10 + iter);
	simulate(arg_hz);
	
	// reset device
	HANDLE_ERROR(hipDeviceReset());
}